#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_DIM 1024

// This is the code from the book but I couldn't get this to run faster even
// with occupancy calculator L1 throughput is dramatically increased though
__global__ void SharedMemoryReduction(float *input, float *output) {
  __shared__ float input_s[BLOCK_DIM];
  unsigned int t = threadIdx.x;
  input_s[t] = input[t] + input[t + BLOCK_DIM];
  for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
    __syncthreads();
    if (threadIdx.x < stride) {
      input_s[t] += input_s[t + stride];
    }
  }

  if (threadIdx.x == 0) {
    *output = input_s[0];
  }
}

int main() {
  // Size of the input data
  const int size = 2048;
  const int bytes = size * sizeof(float);

  // Allocate memory for input and output on host
  float *h_input = new float[size];
  float *h_output = new float;

  // Initialize input data on host
  for (int i = 0; i < size; i++) {
    h_input[i] = 1.0f; // Example: Initialize all elements to 1
  }

  // Allocate memory for input and output on device
  float *d_input;
  float *d_output;
  hipMalloc(&d_input, bytes);
  hipMalloc(&d_output, sizeof(float));

  // Copy data from host to device
  hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

  // Launch the kernel
  SharedMemoryReduction<<<1, size / 2>>>(d_input, d_output);

  // Copy result back to host
  hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

  // Print the result
  std::cout << "Sum is " << *h_output << std::endl;

  // Cleanup
  delete[] h_input;
  delete h_output;
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
